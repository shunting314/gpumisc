/*
 * Need add -lcublas to nvcc command.
 */

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <hipblas.h>

void matmul_cpu(float* A, float *B, float *C, int N) {
  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      float acc = 0.0f;
      for (int k = 0; k < N; ++k) {
        acc += A[i * N + k] * B[k * N + j];
      }
      C[i * N + j] = acc;
    }
  }
}

#define THREAD_PER_ELEM 0
#define THREAD_PER_ROW 1
#define THREAD_PER_COL 2
#define CHOICE THREAD_PER_COL

__global__ void matmul_cuda_kernel(float* A, float* B, float* C, int N) {
  int colIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int rowIdx = blockIdx.y * blockDim.y + threadIdx.y;

  if (colIdx < N && rowIdx < N) {
    float acc = 0.0f;
    for (int k = 0; k < N; ++k) {
      acc += A[rowIdx * N + k] * B[k * N + colIdx];
    }
    C[rowIdx * N + colIdx] = acc;
  }
}

__global__ void matmul_cuda_kernel_per_row(float* A, float* B, float* C, int N) {
  int rowIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (rowIdx < N) {
    for (int colIdx = 0; colIdx < N; ++colIdx) {
      float acc = 0.0f;
      for (int k = 0; k < N; ++k) {
        acc += A[rowIdx * N + k] * B[k * N + colIdx];
      }
      C[rowIdx * N + colIdx] = acc;
    }
  }
}

__global__ void matmul_cuda_kernel_per_col(float* A, float* B, float* C, int N) {
  int colIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (colIdx < N) {
    for (int rowIdx = 0; rowIdx < N; ++rowIdx) {
      float acc = 0.0f;
      for (int k = 0; k < N; ++k) {
        acc += A[rowIdx * N + k] * B[k * N + colIdx];
      }
      C[rowIdx * N + colIdx] = acc;
    }
  }
}

void matmul_cuda(float *h_A, float* h_B, float *h_C, int N) {
  float* d_A, *d_B, *d_C;
  int nbytes = N * N * sizeof(float);
  hipMalloc(&d_A, nbytes);
  hipMalloc(&d_B, nbytes);
  hipMalloc(&d_C, nbytes);
  hipMemcpy(d_A, h_A, nbytes, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, nbytes, hipMemcpyHostToDevice);

  #if CHOICE == THREAD_PER_ELEM
  int blksize = 16;
  int nblk = (N + blksize - 1) / blksize;
  matmul_cuda_kernel<<<dim3(nblk, nblk), dim3(blksize, blksize)>>>(d_A, d_B, d_C, N);
  #endif

  #if CHOICE == THREAD_PER_ROW
  int blksize = 64;
  int nblk = (N + blksize - 1) / blksize;
  matmul_cuda_kernel_per_row<<<nblk, blksize>>>(d_A, d_B, d_C, N);
  #endif

  #if CHOICE == THREAD_PER_COL
  int blksize = 64;
  int nblk = (N + blksize - 1) / blksize;
  matmul_cuda_kernel_per_col<<<nblk, blksize>>>(d_A, d_B, d_C, N);
  #endif

  hipMemcpy(h_C, d_C, nbytes, hipMemcpyDeviceToHost);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

void transpose(float* mat, int N) {
  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < i; ++j) {
      float tmp = mat[i * N + j];
      mat[i * N + j] = mat[j * N + i];
      mat[j * N + i] = tmp;
    }
  }
}

void matmul_cublas(float* h_A, float* h_B, float* h_C, int N) {
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  float* d_A, *d_B, *d_C;
  float alpha = 1.0f, beta = 0.0f;
  int nbytes = N * N * sizeof(float);
  hipMalloc(&d_A, nbytes);
  hipMalloc(&d_B, nbytes);
  hipMalloc(&d_C, nbytes);
  hipMemcpy(d_A, h_A, nbytes, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, nbytes, hipMemcpyHostToDevice);

  // By default, cublas matrix is column major. Need handle the transpose of input/output when needed.
  hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N);

  hipMemcpy(h_C, d_C, nbytes, hipMemcpyDeviceToHost);
  transpose(h_C, N);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipblasDestroy(handle);
}

int calc_nfail(float* ref, float* act, int S) {
  int nfail = 0;
  float rtol = 1e-5;
  float atol = 1e-8;
  for (int i = 0; i < S; ++i) {
    bool pass = (fabs(ref[i] - act[i]) <= atol + rtol * fabs(act[i]));
    if (!pass) {
      ++nfail;
    }
  }
  return nfail;
}

void printMat(const char* prompt, float* mat, int N) {
  printf("%s\n", prompt);
  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      printf(" %6f", mat[i * N + j]);
    }
    printf("\n");
  }
}

int main(void) {
  const int SIZE = 512;
  // const int SIZE = 4;
  float* h_A = (float*) malloc(SIZE * SIZE * sizeof(float));
  float* h_B = (float*) malloc(SIZE * SIZE * sizeof(float));
  float* h_C_cpuref = (float*) malloc(SIZE * SIZE * sizeof(float));
  float* h_C_cublas = (float*) malloc(SIZE * SIZE * sizeof(float));
  float* h_C_cuda = (float*) malloc(SIZE * SIZE * sizeof(float));

  for (int i = 0; i < SIZE; ++i) {
    for (int j = 0; j < SIZE; ++j) {
      h_A[i * SIZE + j] = float(rand()) / RAND_MAX;
      h_B[i * SIZE + j] = float(rand()) / RAND_MAX;
    }
  }

  matmul_cpu(h_A, h_B, h_C_cpuref, SIZE);
  matmul_cuda(h_A, h_B, h_C_cuda, SIZE);
  matmul_cublas(h_A, h_B, h_C_cublas, SIZE);

  if (SIZE < 10) {
    printMat("A:", h_A, SIZE);
    printMat("B:", h_B, SIZE);
    printMat("C_ref:", h_C_cpuref, SIZE);
    printMat("C_cuda:", h_C_cuda, SIZE);
    printMat("C_cublas:", h_C_cublas, SIZE);
  }

  int nfail = calc_nfail(h_C_cpuref, h_C_cuda, SIZE * SIZE);
  if (nfail > 0) {
    fprintf(stderr, "CUDA implementation does not match with cpu. %d mismatch\n", nfail);
    return -1;
  }

  nfail = calc_nfail(h_C_cpuref, h_C_cublas, SIZE * SIZE);
  if (nfail > 0) {
    fprintf(stderr, "cublass implementation does not match with cpu. %d mismatch\n", nfail);
    return -1;
  }

  free(h_A);
  free(h_B);
  free(h_C_cpuref);
  free(h_C_cublas);
  free(h_C_cuda);
  printf("bye\n");
  return 0;
}
