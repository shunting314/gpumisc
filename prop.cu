#include "hip/hip_runtime.h"
#include <stdio.h>

void showDeviceProperties(int devid) {
  printf("Device %d properties:\n", devid);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, devid);
  printf("  name: '%s'\n", prop.name);
  printf("  #SM: %d\n", prop.multiProcessorCount);
  printf("  #global mem: %.3fGB (%.3fGiB)\n", (double) prop.totalGlobalMem / 1e9, (double) prop.totalGlobalMem / (1 << 30));
  printf("  #L2$: %.3fMB (%.3fMiB)\n", (double) prop.l2CacheSize / 1e6, (double) prop.l2CacheSize / (1 << 20));
  printf("  maxThreadsPerBlock %d\n", prop.maxThreadsPerBlock);
  printf("  clockRate %d\n", prop.clockRate); // core clock rate
  for (int i = 0; i < 3; ++i) {
    printf("  maxThreadsDim[%d] %d\n", i, prop.maxThreadsDim[i]);
  }
  for (int i = 0; i < 3; ++i) {
    printf("  maxGridSize[%d] %d\n", i, prop.maxGridSize[i]);
  }
  printf("  regsPerBlock %d\n", prop.regsPerBlock);
  printf("  regsPerMultiprocessor %d\n", prop.regsPerMultiprocessor);
  printf("  warpSize %d\n", prop.warpSize);
  printf("  sharedMemPerBlock %d\n", prop.sharedMemPerBlock);
  printf("  sharedMemPerMultiprocessor %d\n", prop.sharedMemPerMultiprocessor);
}

int main(void) {
  int ngpu = -1;
  hipGetDeviceCount(&ngpu);
  printf("#GPU %d\n", ngpu);

  // show device properties for each available devices
  for (int i = 0; i < ngpu; ++i) {
    showDeviceProperties(i);
  }
  return 0;
}
